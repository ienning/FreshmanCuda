#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <freshman.h>
#include <hip/device_functions.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// CPU
int recursiveReduce(int* data, int const size)
{
    if (size == 1)
    {
        return data[0];
    }

    int stride = size / 2;
    if (size % 2 == 1)
    {
        for (size_t i = 0; i < stride; i++)
        {
            data[i] += data[i + stride];
        }
        data[0] += data[size - 1];
    }
    else
    {
        for (size_t i = 0; i < stride; i++)
        {
            data[i] += data[i + stride];
        }
    }
    return recursiveReduce(data, stride);
}

//GPU
__global__ void warmup(int* g_idata, int* g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;

    if (tid >= n)
    {
        return;
    }

    int* idata = g_idata + blockIdx.x * blockDim.x;

    for (size_t stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2*stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        g_odata[blockIdx.x] = idata[0];
    }
}

__global__ void reduceNeighbored(int* g_idata, int* g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;

    if (tid >= n)
    {
        return;
    }

    int* idata = g_idata + blockIdx.x * blockDim.x;

    for (size_t stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2*stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        g_odata[blockIdx.x] = idata[0];
    }

}

__global__ void reduceNeighboredLess(int* g_idata, int* g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    int* idata = g_idata + blockIdx.x * blockDim.x;
    if (idx > n)
    {
        return;
    }

    for (size_t stride = 1; stride < blockDim.x; stride *= 2)
    {
        int index = 2 * stride * tid;
        if (index < blockDim.x)
        {
            idata[index] += idata[index + stride];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        g_odata[blockIdx.x] = idata[0];
    }
}

__global__ void reduceInterleaved(int* g_idata, int* g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int* idata = g_idata + blockIdx.x * blockDim.x;
    if (idx >= n)
    {
        return;
    }

    int stride = blockDim.x / 2;
    for (; stride > 0; stride>>=1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        g_odata[blockIdx.x] = idata[0];
    }
}

int main(int argc, char** argv)
{
	initDevice(0);
	
	bool bResult = false;
	//initialization

	int size = 1 << 28;
	printf("	with array size %d  ", size);

	//execution configuration
	int blocksize = 1024;
	if (argc > 1)
	{
		blocksize = atoi(argv[1]);
	}
	dim3 block(blocksize, 1);
	dim3 grid((size - 1) / block.x + 1, 1);
	printf("grid %d block %d \n", grid.x, block.x);

	//allocate host memory
	size_t bytes = size * sizeof(int);
	int *idata_host = (int*)malloc(bytes);
	int *odata_host = (int*)malloc(grid.x * sizeof(int));
	int * tmp = (int*)malloc(bytes);

	//initialize the array
	initialData_int(idata_host, size);

	memcpy(tmp, idata_host, bytes);
	double iStart, iElaps;
	int gpu_sum = 0;

	// device memory
	int * idata_dev = NULL;
	int * odata_dev = NULL;
	CHECK(hipMalloc((void**)&idata_dev, bytes));
	CHECK(hipMalloc((void**)&odata_dev, grid.x * sizeof(int)));

	//cpu reduction
	int cpu_sum = 0;
	iStart = cpuSecond();
	//cpu_sum = recursiveReduce(tmp, size);
	for (int i = 0; i < size; i++)
		cpu_sum += tmp[i];
	printf("cpu sum:%d \n", cpu_sum);
	iElaps = cpuSecond() - iStart;
	printf("cpu reduce                 elapsed %lf ms cpu_sum: %d\n", iElaps, cpu_sum);


	//kernel 1:reduceNeighbored

	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	iStart = cpuSecond();
	warmup <<<grid, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];
	printf("gpu warmup                 elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
		iElaps, gpu_sum, grid.x, block.x);

	//kernel 1:reduceNeighbored

	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	iStart = cpuSecond();
	reduceNeighbored << <grid, block >> >(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];
	printf("gpu reduceNeighbored       elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
		iElaps, gpu_sum, grid.x, block.x);

	//kernel 2:reduceNeighboredLess

	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	iStart = cpuSecond();
	reduceNeighboredLess <<<grid, block>>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];
	printf("gpu reduceNeighboredLess   elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
		iElaps, gpu_sum, grid.x, block.x);

	//kernel 3:reduceInterleaved
	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	iStart = cpuSecond();
	reduceInterleaved << <grid, block >> >(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];
	printf("gpu reduceInterleaved      elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
		iElaps, gpu_sum, grid.x, block.x);
	// free host memory

	free(idata_host);
	free(odata_host);
	CHECK(hipFree(idata_dev));
	CHECK(hipFree(odata_dev));

	//reset device
	hipDeviceReset();

	//check the results
	if (gpu_sum == cpu_sum)
	{
		printf("Test success!\n");
	}
	return EXIT_SUCCESS;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
